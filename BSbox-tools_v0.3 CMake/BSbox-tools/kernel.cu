﻿////////////////////////////////////////////////////////////////////////////
//
// Copyright @2017-2023 Dusan and Iliya.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////
//Example for using GPU BoolSPLG library procedures for Boolean function  
////////////////////////////////////////////////////////////////////////////

//System includes
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <string>
#include <sstream>
#include <fstream>
#include <time.h>

// CUDA runtime
#include "hip/hip_runtime.h"
#include ""

//Main Library header file
#include <BoolSPLG/BoolSPLG_v03.cuh> //#include "BoolSPLG_v0.cuh" 

//Output file 
std::ofstream myfile;

//Declaration strings for input and output files names
std::string infile_exampl = "infile_exampl";
std::string outfile_exampl = "outfile_exampl";

//@Global Time computation variables
double elapsedTime_computation_part = 0, elapsedTime_computation = 0;

//@@Global Size variable
int sizeB = 0;

//@@Global Bool variables
int sizeBool;

//Declaration host vectors for Boolean function
int* host_Bool_TT, * host_Bool_PTT, * host_GPUbool_rez, * walshvec_cpu, * rf_cpu, * anf_cpu;

//Declaration device vector for Boolean function
int* device_Bool, * device_Bool_TT, * device_Bool_PTT, * device_Bool_rez;

//@Declaration device vectors for Boolean Bitwise (ANF) function
unsigned long long int* device_NumIntVecTT, * device_NumIntVecANF;
int* device_max_values_AD;

//host vectors for S-box
int* LAT, * ACT, * ANF, * CF, * DDT, * PTT, * TT, * t, * WHT, * AC_all, * ANF_S, * DEG_ALL, * delta_ALL, * binary_num;

//@Declaration device S-box vectors
int* device_Sbox, * device_CF, * device_LAT, * device_ACT, * device_ANF, * device_DDT;

//@Declaration host vectors for Boolean Bitwise (ANF) function
unsigned long long int* NumIntVecTT, * NumIntVecANF; 
int* host_max_values_AD;

//@Declaration of host - device vector for Bitwise ANF computation (S-box)
unsigned long long int* device_NumIntVecCF, * host_NumIntVecCF;

//@Declaration of host - device vector 
int* host_CF, * host_max_values, * device_Vec_max_values;

//@@S-box variables
int sizeSbox, binary = 0;

int* BoolElemet, * SboxElemet, * CPU_CF;

//@@Variable borders
int nl_gpuBorder = 0, AC_gpuBorder = 0, deg_gpuBorder = 0, delta_gpuBorder = 0;
int nl_cpuBorder = 0, AC_cpuBorder = 0, deg_cpuBorder = 0, delta_cpuBorder = 0;

//@@Global variable
int choice1, choice2;

//function menu
void menu();
void mainMenu();

void menuCPU();
void mainMenuCPU();

void CUDA_Properties();
void CUDA_PropertiesMenu();

void CPU_boolMenu();
void CPU_bool();

void GPU_boolMenu();
void GPU_bool();

void CPU_SboxMenu();
void CPU_Sbox();

void GPU_SboxMenu();
void GPU_Sbox();

void findCPU_boolMenu();
void findCPU_bool();

void findGPU_boolMenu();
void findGPU_bool();

void findCPU_sboxMenu();
void findCPU_sbox();

void findGPU_sboxMenu();
void findGPU_sbox();

//Help Heder file - Input Output file
#include "InputOutputFiles.h"

//Help Header file - Time functions
#include "TimeHeader.h"

////Help Heder file - CPU computing boolean functions properties
//#include "func_Boolean_CPU.h"
//#include "funct_Sbox_CPU.h"
//
////Heder file 2D DynamicArray
////#include "2D_DynamicArray.h"

//Help Heder file boolean computation functions
#include "HelpBoolFunctionCPU.h"
#include "HelpBoolFunctionGPU.h"


#include "HelpFindBoolFunctionGPU.h"
#include "HelpFindBoolFunctionCPU.h"

//Help Heder file S-box computation functions
#include "HelpSboxFunctionCPU.h"
#include "HelpSboxFunctionGPU.h"

#include "HelpFindSboxFunctionCPU.h"
#include "HelpFindSboxFunctionGPU.h"

//Header file that contain S-box generation
#include "GenBoolean.h"
#include "GenSboxes.h"

//Menu Heder CPU - GPU Bool Properties
#include "ComputePropMenuBoolCPU.h"
#include "ComputePropMenuBoolGPU.h"


//Menu Heder CPU - GPU S-box Properties
#include "ComputePropMenuSboxCPU.h"
#include "ComputePropMenuSboxGPU.h"

// === Menu Heders files ===
#include "MainMenuCPU.h"
#include "MainMenu.h"

//Menu Heders CUDA Properties
#include "CUDA_Properties_menu.h"

#include "FindMenuBoolCPU.h"
#include "FindMenuBoolGPU.h"
#include "FindMenuSboxCPU.h"
#include "FindMenuSboxGPU.h"

//Help Heder file - Check for hardware Requires
#include "CheckRequires.h"

using namespace std;

int main()
{
	cout << "==========================================================";
	printf("\nBSbox-tools, Software Boolean S-box Cryptographic Properties Tool for GPUs.\n");
	printf("\n   Current release: v0.3 \n");

	//Function create Output files
	CreateOutputFile();

	cout << "\n==========================================================\n";
	//Function Check Hardware Requires, and to switch to CPU menu if does not have GPU
	BoolSPLGMinimalRequires1();
	cout << "\n==========================================================\n";

	//Function Open Main Menu
	menu();

	return 0;
}